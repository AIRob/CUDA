

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

typedef enum TARGET {HOST, DEVICE} TARGET;
typedef enum MEMTYPE {NORMAL, PINNED} MEMTYPE;

typedef struct {
    int width;
    int height;
    float *elements;
} Matrix;

__global__ void sgemm(Matrix A, Matrix B, Matrix C, 
                      const float alpha, const float beta, 
                      const int width, const int height) {
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
    int idx = idx_y * width + idx_x;
    
    if (idx_x >= width || idx_y >= height)
        return;
    
    float value = 0.f;
    for (int e = 0; e < width; e++)
        value = alpha * A.elements[idx_y * width + e] * B.elements[e * width + idx_x];
    C.elements[idx] = value + beta * C.elements[idx];
}

void InitMatrix(Matrix &mat, const int width, const int height, TARGET target = HOST, MEMTYPE memtype = NORMAL);

int main(int argv, char* argc[]) {
    Matrix A, B, C;
    Matrix dA, dB, dC;
    const float alpha = 2.f;
    const float beta = .5f;
    const int width = 2048;
    const int height = 2048;
    float elapsed_gpu;
    double elapsed_cpu;
    
    // Select Host memory type (NORMAL, PINNED)
    MEMTYPE memtype = PINNED;
    
    // CUDA Event Create to estimate elased time
    hipEvent_t start, stop;
    struct timespec begin, finish;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Initialize host matrix
    InitMatrix(A, width, height, HOST, memtype);
    InitMatrix(B, width, height, HOST, memtype);
    InitMatrix(C, width, height, HOST, memtype);

    // CUDA Memory Initialize
    InitMatrix(dA, width, height, DEVICE);
    InitMatrix(dB, width, height, DEVICE);
    InitMatrix(dC, width, height, DEVICE);
    
    // CUDA Operation
    hipEventRecord(start, 0);
    clock_gettime(CLOCK_MONOTONIC, &begin);
    
    // Copy host data to the device (CUDA global memory)
    // TODO: Write Asynchronous CUDA Memcpy API (gpu -> cpu)
    
    //////////////
    
    // Launch GPU Kernel
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    sgemm<<<gridDim, blockDim>>>(dA, dB, dC, alpha, beta, width, height);
    
    // Copy computation result from the Device the host memory
    // TODO: Write Asynchronous CUDA Memcpy API (cpu -> gpu)
    
    //////////////
    clock_gettime(CLOCK_MONOTONIC, &finish);
    hipEventRecord(stop, 0);
    
    // Estimate CUDA operation time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&elapsed_gpu, start, stop);
    printf("SGEMM CUDA Elapsed time: %f ms\n", elapsed_gpu);
    elapsed_cpu = (finish.tv_sec - begin.tv_sec);
    elapsed_cpu += (finish.tv_nsec - begin.tv_nsec) / 1000000000.0;
    printf("Host time: %f ms\n", elapsed_cpu * 1000);
    
    // finalize CUDA event
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    // Finalize
    hipFree(dA.elements);
    hipFree(dB.elements);
    hipFree(dC.elements);
    
    if (memtype == NORMAL) {
        free(A.elements);
        free(B.elements);
        free(C.elements);
    }
    else {
        // TODO: Write pinned memory free API

        /////////////
    }
    
    return 0;
}

void InitMatrix(Matrix &mat, const int width, const int height, TARGET target, MEMTYPE memtype) {
    mat.width = width;
    mat.height = height;
    
    if (target == DEVICE) {
        hipMalloc((void**)&mat.elements, width * height * sizeof(float));
    }
    else {
        if (memtype == NORMAL)
            mat.elements = (float*)malloc(width * height * sizeof(float));
        else
            // TODO: Write pinned memory allocation API
            
            /////////////////////
    
        for (int row = 0; row < height; row++) {
            for (int col = 0; col < width; col++) {
                mat.elements[row * width + col] = row * width + col * 0.001;
            }
        }
    }
}