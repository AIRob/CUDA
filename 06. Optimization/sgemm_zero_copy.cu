#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime_api.h>

typedef enum TARGET {HOST, DEVICE} TARGET;

typedef struct {
    int width;
    int height;
    float *elements;
} Matrix;

__global__ void sgemm(Matrix A, Matrix B, Matrix C, 
                      const float alpha, const float beta, 
                      const int width, const int height) {
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
    int idx = idx_y * width + idx_x;
    
    if (idx_x >= width || idx_y >= height)
        return;
    
    float value = 0.f;
    for (int e = 0; e < width; e++)
        value = alpha * A.elements[idx_y * width + e] * B.elements[e * width + idx_x];
    C.elements[idx] = value + beta * C.elements[idx];
}

void InitMatrix(Matrix &mat, const int width, const int height, TARGET target = HOST);

int main(int argv, char* argc[]) {
    Matrix A, B, C;
    Matrix dA, dB, dC;
    const float alpha = 2.f;
    const float beta = .5f;
    const int width = 2048;
    const int height = 2048;
    float elapsed_gpu;
    double elapsed_cpu;
    
    // CUDA Event Create to estimate elased time
    hipEvent_t start, stop;
    struct timespec begin, finish;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Initialize host matrix
    InitMatrix(A, width, height);
    InitMatrix(B, width, height);
    InitMatrix(C, width, height);
    
    // CUDA Memory Initialize
    // TODO: Write device memory pointer obtaining code from host pointer
    
    
    ///////////////
    
    // CUDA Operation
    hipEventRecord(start, 0);
    clock_gettime(CLOCK_MONOTONIC, &begin);
    
    // Copy host data to the device (CUDA global memory)
    //hipMemcpy(dA.elements, A.elements, width * height * sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(dB.elements, B.elements, width * height * sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(dC.elements, C.elements, width * height * sizeof(float), hipMemcpyHostToDevice);
    
    // Launch GPU Kernel
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    sgemm<<<gridDim, blockDim>>>(dA, dB, dC, alpha, beta, width, height);
    
    // Copy computation result from the Device the host memory
    //hipMemcpy(C.elements, dC.elements, width * height * sizeof(float), hipMemcpyDeviceToHost);
    
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &finish);
    
    // Estimate CUDA operation time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&elapsed_gpu, start, stop);
    printf("SGEMM CUDA Elapsed time: %f ms\n", elapsed_gpu);
    elapsed_cpu = (finish.tv_sec - begin.tv_sec);
    elapsed_cpu += (finish.tv_nsec - begin.tv_nsec) / 1000000000.0;
    printf("Host time: %f ms\n", elapsed_cpu * 1000);
    
    // finalize CUDA event
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    // Finalize
    //hipFree(dA.elements);
    //hipFree(dB.elements);
    //hipFree(dC.elements);
    
    hipHostFree(A.elements);
    hipHostFree(B.elements);
    hipHostFree(C.elements);
    
    return 0;
}

void InitMatrix(Matrix &mat, const int width, const int height, TARGET target) {
    mat.width = width;
    mat.height = height;
    
    if (target == DEVICE) {
        hipMalloc((void**)&mat.elements, width * height * sizeof(float));
    }
    else {
        // TODO: write pinned memory allocation code (mapped mode)
        
        ///////////////////

        for (int row = 0; row < height; row++) {
            for (int col = 0; col < width; col++) {
                mat.elements[row * width + col] = row * width + col * 0.001;
            }
        }
    }
}