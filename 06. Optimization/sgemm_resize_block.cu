#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

typedef enum TARGET {HOST, DEVICE} TARGET;

typedef struct {
    int width;
    int height;
    float *elements;
} Matrix;

__global__ void sgemm(Matrix A, Matrix B, Matrix C, 
                      const float alpha, const float beta, 
                      const int width, const int height) {
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
    int idx = idx_y * width + idx_x;
    
    if (idx_x >= width || idx_y >= height)
        return;
    
    // TODO: Copy sgemm code from above you write
}

void InitMatrix(Matrix &mat, const int width, const int height, TARGET target = HOST);

int main(int argv, char* argc[]) {
    Matrix A, B, C_host, C_device;
    Matrix dA, dB, dC;
    const float alpha = 2.f;
    const float beta = .5f;
    const int width = 2048;
    const int height = 2048;
    float elapsed_gpu;
    double elapsed_cpu;
    
    // CUDA Event Create to estimate elased time
    hipEvent_t start, stop;
    struct timespec begin, finish;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Initialize host matrix
    InitMatrix(A, width, height);
    InitMatrix(B, width, height);
    InitMatrix(C_device, width, height);

    // CUDA Memory Initialize
    InitMatrix(dA, width, height, DEVICE);
    InitMatrix(dB, width, height, DEVICE);
    InitMatrix(dC, width, height, DEVICE);
    
    // CUDA Operation
    hipEventRecord(start, 0);
    clock_gettime(CLOCK_MONOTONIC, &begin);
    
    // Copy host data to the device (CUDA global memory)
    hipMemcpy(dA.elements, A.elements, width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB.elements, B.elements, width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dC.elements, C_device.elements, width * height * sizeof(float), hipMemcpyHostToDevice);
    
    // Launch GPU Kernel
    // TODO: Define your block size and check the performance
    dim3 blockDim(BLOCK_DIM, BLOCK_DIM);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    sgemm<<<gridDim, blockDim>>>(dA, dB, dC, alpha, beta, width, height);
    
    // Copy computation result from the Device the host memory
    hipMemcpy(C_device.elements, dC.elements, width * height * sizeof(float), hipMemcpyDeviceToHost);
    clock_gettime(CLOCK_MONOTONIC, &finish);
    hipEventRecord(stop, 0);
    
    // Estimate CUDA operation time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&elapsed_gpu, start, stop);
    printf("SGEMM CUDA Elapsed time: %f ms\n", elapsed_gpu);
    elapsed_cpu = (finish.tv_sec - begin.tv_sec);
    elapsed_cpu += (finish.tv_nsec - begin.tv_nsec) / 1000000000.0;
    printf("Host time: %f ms\n", elapsed_cpu * 1000);
    
    // finalize CUDA event
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    // Finalize
    hipFree(dA.elements);
    hipFree(dB.elements);
    hipFree(dC.elements);
    
    free(A.elements);
    free(B.elements);
    //free(C_host.elements);
    free(C_device.elements);
    
    return 0;
}

void InitMatrix(Matrix &mat, const int width, const int height, TARGET target) {
    mat.width = width;
    mat.height = height;
    
    if (target == DEVICE) {
        hipMalloc((void**)&mat.elements, width * height * sizeof(float));
    }
    else {
        mat.elements = (float*)malloc(width * height * sizeof(float));
    
        for (int row = 0; row < height; row++) {
            for (int col = 0; col < width; col++) {
                mat.elements[row * width + col] = row * width + col * 0.001;
            }
        }
    }
}