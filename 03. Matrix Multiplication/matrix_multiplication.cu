#include "hip/hip_runtime.h"

#include <stdio.h>
#include "matrix_multiplication.h"
#include "util.h"

__global__
void d_matrix_multiplication(Matrix C, Matrix A, Matrix B) {
    // TODO
    // Matrix Multiplication 연산 GPU 코드를 작성하세요.
    // 필요하다면 matrix_multiplication_host.cc 파일을 참고하세요.
    
}

void matrix_multiplication(Matrix &C, Matrix A, Matrix B) {
    int size;
    
    ////////////////////////////////
    // CUDA Event Create to estimate elased time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // CUDA Operation
    hipEventRecord(start, 0);
    /////////////////////////////////
    
    // Create GPU memory
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size = A.width * A.height * sizeof(int);
    hipMalloc((void**)&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    
    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(int);
    hipMalloc((void**)&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
    
    Matrix d_C;
    d_C.height = C.height; d_C.width = C.width;
    size = C.width * C.height * sizeof(int);
    hipMalloc((void**)&d_C.elements, size);
    
    // Kernel launch
    // TODO: call CUDA Kernel
    dim3 blockDim(16, 16);
    dim3 gridDim((d_C.width + blockDim.x - 1) / blockDim.x, (d_C.height + blockDim.y - 1) / blockDim.y);
    ...

    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
    
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
    
    /////////////////////////////////
    // Estimate CUDA operation time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("CUDA Elapsed time: %f ms\n", elapsedTime);
    
    // finalize CUDA event
    hipEventDestroy(start);
    hipEventDestroy(stop);
    /////////////////////////////////
}

int main() {
    int width_a = 256;
    int height_a = 128;
    int height_b = 256;
    int width_b = height_a;
    
    Matrix A, B, C, C_cuda;
    
    init_matrix(&A, width_a, height_a, 1);
    init_matrix(&B, width_b, height_b, 2);
    init_matrix(&C, A.height, B.width, 0);
    init_matrix(&C_cuda, A.height, B.width, 0);
    
    // Matrix Multiplication
    matrix_multiplication(C_cuda, A, B);
    matrix_multiplication_host(C, A, B);
    
    // Check results
    check_result(C, C_cuda);
    
    free(A.elements);
    free(B.elements);
    free(C.elements);
    free(C_cuda.elements);
     
    return 0;
}