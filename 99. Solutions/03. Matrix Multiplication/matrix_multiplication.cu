#include "hip/hip_runtime.h"

#include <stdio.h>
#include "matrix_multiplication.h"
#include "util.h"

__global__
void d_matrix_multiplication(Matrix C, Matrix A, Matrix B) {
    // TODO
    // Matrix Multiplication 연산 GPU 코드를 작성하세요.
    // 필요하다면 matrix_multiplication_host.cc 파일을 참고하세요.
    
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
    
    if (idx_x < C.width) {
        int value = 0.f;
        for (int e = 0; e < A.width; e++) {
            value += A.elements[idx_y * A.width + e] * B.elements[e * B.width + idx_x];
        }
        C.elements[idx_y * C.width + idx_x] = value;
    }
}

void matrix_multiplication(Matrix &C, Matrix A, Matrix B) {
    int size;
    
    ////////////////////////////////
    // CUDA Event Create to estimate elased time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // CUDA Operation
    hipEventRecord(start, 0);
    /////////////////////////////////
    
    // Create GPU memory
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size = A.width * A.height * sizeof(int);
    hipMalloc((void**)&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    
    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(int);
    hipMalloc((void**)&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
    
    Matrix d_C;
    d_C.height = C.height; d_C.width = C.width;
    size = C.width * C.height * sizeof(int);
    hipMalloc((void**)&d_C.elements, size);
    
    // Kernel launch
    // TODO: CUDA 함수를 호출하세요
    dim3 blockDim(16, 16);
    dim3 gridDim((d_C.width + blockDim.x - 1) / blockDim.x, (d_C.height + blockDim.y - 1) / blockDim.y);
    d_matrix_multiplication<<<gridDim, blockDim>>>(d_C, d_A, d_B);

    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
    
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
    
    /////////////////////////////////
    // Estimate CUDA operation time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("CUDA Elapsed time: %f ms\n", elapsedTime);
    
    // finalize CUDA event
    hipEventDestroy(start);
    hipEventDestroy(stop);
    /////////////////////////////////
}

int main() {
    int width_a = 256;
    int height_a = 128;
    int height_b = 256;
    int width_b = height_a;
    
    Matrix A, B, C, C_cuda;
    
    init_matrix(&A, width_a, height_a, 1);
    init_matrix(&B, width_b, height_b, 2);
    init_matrix(&C, A.height, B.width, 0);
    init_matrix(&C_cuda, A.height, B.width, 0);
    
    // Matrix Multiplication
    matrix_multiplication(C_cuda, A, B);
    matrix_multiplication_host(C, A, B);
    
    // Check results
    check_result(C, C_cuda);
    
    free(A.elements);
    free(B.elements);
    free(C.elements);
    free(C_cuda.elements);
     
    return 0;
}